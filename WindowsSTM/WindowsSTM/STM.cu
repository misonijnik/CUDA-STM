#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdint.h>
#include "STM.cuh"
#include "helper.cuh"
#include "CUDAStructures.cuh"

__host__ int hey()
{
	hipError_t error = hipDeviceSynchronize();
	cudaCheckError();
	int deviceCount;
	hipGetDeviceCount(&deviceCount);
	cudaCheckError();
	int device;
	for (device = 0; device < deviceCount; ++device)
	{
		hipDeviceProp_t deviceProp;
		hipGetDeviceProperties(&deviceProp, device);
		cudaCheckError();
		printf("Device %d has compute capability %d.%d.%lu.\n", device, deviceProp.major, deviceProp.minor, deviceProp.totalGlobalMem);
	}
	return 0;
}

__host__ int hey3()
{
	size_t size;
	hipDeviceGetLimit(&size, hipLimitMallocHeapSize);
	//hipDeviceSetLimit(hipLimitMallocHeapSize, size * 2);
	printf("%u.\n", size);
	double* ptr;
	hipMalloc((void**)&ptr, sizeof(double)*100);
	hipDeviceSynchronize();
	cudaCheckError();
	double* val = (double*)malloc(sizeof(double)*100);
	cudaCheckError();
	val[0] = 0;
	val[1] = 0;
	//hipMemcpy(ptr, val, sizeof(int)*100, hipMemcpyHostToDevice);
	hipMemset(ptr, 0, sizeof(double) * 100);
	hipDeviceSynchronize();
	cudaCheckError();
	GlobalLockTable g_lock = GlobalLockTable(ptr, sizeof(double), 100, 1);
	cudaCheckError();
	testCorrectSTM<<<100,1024>>>(g_lock, ptr);//todo fix error with more block
	hipDeviceSynchronize();
	cudaCheckError();
	g_lock.Dispose();
	hipMemcpy(val, ptr, sizeof(double)*100, hipMemcpyDeviceToHost);
	printf("%f.\n", (val[0]));
	printf("%f.\n", (val[1]));
	free(val);
	hipFree(val);

	return 0;
}

__host__ int hey2()
{
	CUDAArray<WriteEntry<int> > hmm = CUDAArray<WriteEntry<int> >(10);
	hipDeviceSynchronize();
	cudaCheckError();
	int* ptr;
	hipMalloc((void**)&ptr, sizeof(int)*2);
	int* ptr2 = ptr + 1;
	hipDeviceSynchronize();
	cudaCheckError();
	printf("%lu.\n", (uintptr_t)ptr);
	printf("%lu.\n", (uintptr_t)(ptr2));
	printf("%lu.\n", ((uintptr_t)(ptr2) - (uintptr_t)(ptr)));
	printf("%p.\n", (ptr2));
	printf("%p.\n", (void*)(19662336));
	changeArray<<<1,1>>>(hmm, ptr, 7);
	hipDeviceSynchronize();
	cudaCheckError();

	WriteEntry<int>* entryPtr;
	entryPtr = hmm.GetData();

	cudaCheckError();

	int* tmp = (int*)malloc(sizeof(int));
	*tmp= entryPtr[0].value;
	int intTmp = *tmp;
	printf("Здравствуй, %d мир!\n", intTmp);
	cudaCheckError();

	free(tmp);
	hipFree(ptr);

	return 0;
}

__host__ int testGlt()
{
	int* ptr;
	hipMalloc((void**)&ptr, sizeof(int)*4);
	hipDeviceSynchronize();
	cudaCheckError();
	int* value;
	hipMalloc((void**)&value, sizeof(int));
	GlobalLockTable g_lock = GlobalLockTable(ptr, sizeof(int), 4, 1);
	testGltKernel<<<1,1>>>(g_lock, ptr, value);
	int* val = (int*)malloc(sizeof(int));
	hipMemcpy(val, value, sizeof(int), hipMemcpyDeviceToHost);
	printf("%d.\n", (*val));
	//printf("%lu.\n", g_lock.hash(ptr+1));
	return 0;
}

__global__ void testGltKernel(GlobalLockTable g_lock, int* cudaPtr, int* val)
{
	GlobalLockEntry tmp;
	tmp.entry.locked = 1;

	g_lock.setEntryAt(cudaPtr, tmp);
	tmp.entry.locked = 1;
	g_lock.setEntryAt(cudaPtr+1, tmp);
	*val = g_lock.getEntryAt(cudaPtr).entry.locked;
}

__global__ void testCorrectSTM(GlobalLockTable g_lock, double* cudaPtr)
{
	LocalMetadata<double> local_data = LocalMetadata<double>(&g_lock);
	size_t length = g_lock.getLength();
	size_t count = blockDim.x*gridDim.x;
	unsigned int tmpOne = 0;
	unsigned int tmpTwo = 0;
	unsigned int tmp = uniqueIndex();
	tmp = tmp % 100;
	/*if (uniqueIndex() + 1 > count/2)
	{
		tmpOne++;
	}
	else 
	{
		tmpTwo++;
	}*/
	double val = 0;
	do
	{
		local_data.txStart();
		val = local_data.txRead(cudaPtr + tmp);
		if(local_data.isAborted())
		{
			local_data.releaseLocks();
			continue;
		}
		val++;
		local_data.txWrite(cudaPtr + tmp, val);
		if(local_data.isAborted())
		{
			local_data.releaseLocks();
			continue;
		}

		if(local_data.txValidate())
		{
			local_data.txCommit();
			local_data.releaseLocks();
			break;
		}
	} while (true);
	//printf("thread %u, val %d\n", uniqueIndex(), val);
}

__global__ void changeArray(CUDAArray<WriteEntry<int> > arr, int* ptr, int val)
{
	WriteEntry<int> tmp;
	tmp.cudaPtr = ptr;
	tmp.value = val;

	arr.SetAt(0, tmp);
}
