#include "hip/hip_runtime.h"
#include <stdio.h>
#include "STM.cuh"
#include "helper/helper.cuh"
#include "structures/CUDAStructures.cuh"

__host__ int hey(void)
{
	GlobalLockTable hah = GlobalLockTable(1, 100);

	CUDAArray<WriteEntry> hmm = CUDAArray<WriteEntry>(1, 10);
	int* ptr;
	hipMalloc((void**)&ptr, sizeof(int));
	hipError_t error = hipDeviceSynchronize();
	changeArray<<<1,1>>>(hmm, ptr);
	error = hipDeviceSynchronize();
	WriteEntry* entryPtr = hmm.GetData();
	int* tmp = entryPtr[0].value;
	int intTmp = *tmp;
	printf("Здравствуй, %d мир!\n", intTmp);
	error = hipGetLastError();
	return 0;
}

__global__ void changeArray(CUDAArray<WriteEntry> arr, int* ptr)
{
	WriteEntry tmp;
	tmp.cudaPtr = ptr;
	tmp.value = 6;

	arr.SetAt(0, 0, tmp);
}
